#include "hip/hip_runtime.h"
// main.c
#include <stdio.h>
#include "../include/functions.h"
#include "../include/cuda_kernels.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#include <sys/sysinfo.h>

#define FILE_NAME __FILE__
#define LINE __LINE__
#define BUFFER_SIZE 256



int main(int argc, char** argv) {

    // Selection of best device
    int device = setBestDevice();
    if (device == -1){
        printf("Kernel launch abort\n");
        return -1;
    }

    int k = 10; 
    int metric = 3; // Metric distance
    int exp = 4; // Power for Minkowski distance


    int trainSize = 1000; // Size of the dataset
    int testSize = 100; // Size of the dataset
    int num_features = 10; // Number of features (and classes)
    int num_classes = num_features; // Number of classes
    int mean = 10; // Mean value for class component
    
    // pointer to memory for data and labels
    double *trainData;
    int *trainLabels;
    double *testData;
    int *testLabels;
    
    // Generate training set
    generateData(trainSize, num_features, &trainData, &trainLabels, mean);
    // Generate test set
    generateData(testSize, num_features, &testData, &testLabels, mean);

    // Host memory allocation
    double *distances = (double *)malloc(trainSize * testSize * sizeof(double));
    int *trainIndexes = (int *)malloc(trainSize * testSize * sizeof(int));
    int *predictions = (int *)malloc(testSize * sizeof(int));


    createTrainIndexes(trainIndexes, testSize, trainSize);

    // device memory allocation
    double *d_trainData, *d_testData, *d_distances;
    int *d_trainIndexes, *d_predictions, *d_trainLabels;

    hipMalloc(&d_trainData, trainSize * num_features * sizeof(double));
    hipMalloc(&d_testData, testSize * num_features * sizeof(double));
    hipMalloc(&d_distances, trainSize * testSize * sizeof(double));
    hipMalloc(&d_trainIndexes, trainSize * testSize * sizeof(int));
    hipMalloc(&d_predictions, testSize * sizeof(int));
    hipMalloc(&d_trainLabels, trainSize * sizeof(int));

    hipMemcpy(d_trainData, trainData, trainSize * num_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_testData, testData, testSize * num_features * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_trainIndexes, trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_trainLabels, trainLabels, trainSize * sizeof(int), hipMemcpyHostToDevice);
    

    int dimx = 32;      //default
    int dimy = 32;      //default
    if(argc > 2){
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }
    dim3 block(dimx, dimy);
    dim3 grid((trainSize + block.x-1)/block.x, (testSize + block.y-1)/block.y);

    hipMemset(d_distances, 0, trainSize * testSize * sizeof(double)); // initialize distances matrix with 0
    
    // Set cache configuration for the kernel -> prefer 48KB L1 cache and 16KB shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(knnDistances), hipFuncCachePreferL1);

    double knnDistStart = cpuSecond();
    knnDistances<<< grid, block >>>(d_trainData, d_testData, d_distances, trainSize, testSize, metric, exp, num_features);
    hipDeviceSynchronize();        //forcing synchronous behavior
    double knnDistElaps = cpuSecond() - knnDistStart;
    
    int workers = 10;       // default
    if(argc > 3){
        workers = atoi(argv[3]);
    }

    dim3 gridDim(testSize, 1, 1);   // each thread block is responsible for a row of the distances matrix
    dim3 blockDim(workers, 1, 1);
    int alpha = 2;  // default
    if(argc > 4){
        alpha = atoi(argv[4]);
    }

    int beta = 4;   // default
    if(argc > 5){
        beta = atoi(argv[5]);
    }
    int sharedWorkers = (int)(blockDim.x / alpha);
    int additionalMemory = k * sharedWorkers * (sizeof(double) + sizeof(int));  // blockDim.x/alpha is the number of workers in 2^ iteration (first in shared memory)

    int sharedMemorySize = (k * blockDim.x) * (sizeof(double) + sizeof(int)) + additionalMemory; 

    int index = k * (blockDim.x + sharedWorkers); // starting index for trainIndexes in shared memory 

    double knnSortStart = cpuSecond();
    knnSortPredict<<< gridDim, blockDim, sharedMemorySize>>>(d_distances, trainSize, d_trainIndexes, k, d_predictions, d_trainLabels, index, alpha, beta, num_classes);
    hipDeviceSynchronize();        //forcing synchronous behavior
    double knnSortElaps = cpuSecond() - knnSortStart;

    hipMemcpy(distances, d_distances, trainSize * testSize * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(trainIndexes, d_trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(predictions, d_predictions, testSize * sizeof(int), hipMemcpyDeviceToHost);

    //printDataSet(trainData, trainLabels, trainSize, num_features);

    //printDistances(distances, testSize, trainSize);

    //printTrainIndexes(trainIndexes, testSize, trainSize);


    //check device results
    int errorCount = checkResult(testLabels, predictions, testSize);

    // kernels dimensions
    unsigned int distDim[4] = {grid.x, grid.y, block.x, block.y};
    unsigned int predDim[4] = {gridDim.x, gridDim.y, blockDim.x, blockDim.y};

    // Write results and device info to file
    writeResultsToFile(testLabels, predictions, errorCount, testSize, "par_results_artificial.txt", trainSize, num_features, k, metric, exp, distDim, predDim, workers, alpha, beta, knnDistElaps, knnSortElaps); 
    //writeDeviceInfo("device_info.txt", device);
    writeAllInfoToFile("all_HW_info.txt", device);

    // Free device memory
    hipFree(d_trainData);
    hipFree(d_testData);
    hipFree(d_distances);
    hipFree(d_trainIndexes);
    hipFree(d_predictions);
    hipFree(d_trainLabels);


    // Free host memory
    free(trainData);
    free(trainLabels);
    free(testData);
    free(testLabels);
    free(distances);
    free(trainIndexes);
    free(predictions);


    //reset device
    hipDeviceReset();


    return 0;
}