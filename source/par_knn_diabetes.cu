#include "hip/hip_runtime.h"
#include "../include/cuda_functions.h"
#include "../include/diabetes_functions.h"


int main(int argc, char** argv) {

    // Selection of best device
    int device = setBestDevice();
    if (device == -1){
        printf("Kernel launch abort\n");
        return -1;
    }

    int k = 10; 
    int metric = 3; // Metric distance
    int exp = 4; // Power for Minkowski distance

    Row *dataset;
    int trainSize;
    int testSize;

    // TRAINING DATA
    if (readCSV("../datasets/diabetes_training.csv", &dataset, &trainSize) != 1) {
        printf("Error reading CSV file.\n");
        return 1;
    }

    // Allocate memory for trainData
    double *trainData = (double *)malloc(trainSize * FEATURES * sizeof(double));
    if (trainData == NULL) {
        printf("Error allocating memory.\n");
        free(dataset);
        return 1;
    }

    // Allocate memory for train labels
    int *trainLabels = (int *)malloc(trainSize * sizeof(int));
    if (trainLabels == NULL) {
        printf("Error allocating memory.\n");
        free(dataset);
        free(trainData);
        return 1;
    }

    // Training data extraction
    extractData(dataset, trainData, trainLabels, trainSize);
    //printDataSet(trainData, trainLabels, numRows);

    
    // TEST DATA
    if (readCSV("../datasets/diabetes_testing.csv", &dataset, &testSize) != 1) {
        printf("Error reading CSV file.\n");
        return 1;
    }

    // Allocate memory for testData
    double *testData = (double *)malloc(testSize * FEATURES * sizeof(double));
    if (testData == NULL) {
        printf("Error allocating memory.\n");
        free(dataset);
        return 1;
    }

    // Allocate memory for test labels
    int *testLabels = (int*)malloc(testSize * sizeof(int));
    if (testLabels == NULL) {
        printf("Error allocating memory.\n");
        free(dataset);
        free(testData);
        return 1;
    }

    // Test data extraction
    extractData(dataset, testData, testLabels, testSize);
    //printDataSet(testData, testLabels, testSize);


    double *distances = (double *)malloc(trainSize * testSize * sizeof(double));
    int *trainIndexes = (int *)malloc(trainSize * testSize * sizeof(int));
    int *predictions = (int *)malloc(testSize * sizeof(int));


    createTrainIndexes(trainIndexes, testSize, trainSize);

    // device
    double *d_trainData, *d_testData, *d_distances;
    int *d_trainIndexes, *d_predictions, *d_trainLabels;

    hipMalloc(&d_trainData, trainSize * FEATURES * sizeof(double));
    hipMalloc(&d_testData, testSize * FEATURES * sizeof(double));
    hipMalloc(&d_distances, trainSize * testSize * sizeof(double));
    hipMalloc(&d_trainIndexes, trainSize * testSize * sizeof(int));
    hipMalloc(&d_predictions, testSize * sizeof(int));
    hipMalloc(&d_trainLabels, trainSize * sizeof(int));

    hipMemcpy(d_trainData, trainData, trainSize * FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_testData, testData, testSize * FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_trainIndexes, trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_trainLabels, trainLabels, trainSize * sizeof(int), hipMemcpyHostToDevice);
    

    // Set squared maximum dimensions as default
    int dimx = (int)sqrt(getMaxThreadsPerBlock(device)); 
    int dimy = dimx;

    if(argc > 2){
        if (atoi(argv[1]) * atoi(argv[2]) <= getMaxThreadsPerBlock(device)){
            dimx = atoi(argv[1]);
            dimy = atoi(argv[2]);
        } else {
            printf("Invalid dimensions. Maximum number of threads per block is %d\n", getMaxThreadsPerBlock(device));
            printf("Using default dimensions: %d x %d\n\n", dimx, dimy);
        }
    }
    dim3 block(dimx, dimy);
    dim3 grid((trainSize + block.x-1)/block.x, (testSize + block.y-1)/block.y);

    hipMemset(d_distances, 0, trainSize * testSize * sizeof(double)); // initialize distances matrix with 0
    
    // Set cache configuration for the kernel -> prefer 48KB L1 cache and 16KB shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(knnDistances), hipFuncCachePreferL1);
    
    printf("Executing file: %s\n\n", __FILE__);

    double knnDistStart = cpuSecond();
    knnDistances<<< grid, block >>>(d_trainData, d_testData, d_distances, trainSize, testSize, metric, exp, FEATURES);
    hipDeviceSynchronize();        //forcing synchronous behavior
    double knnDistElaps = cpuSecond() - knnDistStart;
    
    int alpha = 2;  // default
    if(argc > 3){
        if (atoi(argv[3]) >= alpha){
            alpha = atoi(argv[3]);
        } else {
            printf("Invalid alpha value. Using default value: %d\n\n", alpha);
        }
    }

    int beta = 4;   // default
    if(argc > 4){
        if (atoi(argv[4]) >= beta){
            beta = atoi(argv[4]);
        } else {
            printf("Invalid beta value. Using default value: %d\n\n", beta);
        }
    }

    int maxSharedMemory = getSharedMemoryPerBlock(device);
    int itemSize = sizeof(double) + sizeof(int);
    int workers = maxSharedMemory/(k * itemSize * (1.5 + 1/alpha));       // default (maxization of shared memory usage)
    workers = nearestPowerOfTwo(workers);
    if (workers > (int)trainSize/(alpha*k)){
        workers = nearestPowerOfTwo((int)trainSize/(alpha*k));           // new default in case of too many workers (small dataset)
    }

    if(argc > 5){
        if (atoi(argv[5]) < workers){
            workers = atoi(argv[5]);
        } else {
            printf("Invalid workers value. Using default value: %d\n\n", workers);
        }
    }

    dim3 gridDim(testSize, 1, 1);   // each thread block is responsible for a row of the distances matrix
    dim3 blockDim(workers, 1, 1);

    int sharedWorkers = (int)(blockDim.x / alpha);
    int additionalMemory = k * sharedWorkers * (sizeof(double) + sizeof(int));  // blockDim.x/alpha is the number of workers in 2^ iteration (first in shared memory)

    int sharedMemorySize = (k * blockDim.x) * (sizeof(double) + sizeof(int)) + additionalMemory; 

    int index = k * (blockDim.x + sharedWorkers); // starting index for trainIndexes in shared memory 

    double knnStart = cpuSecond();
    knn<<< gridDim, blockDim, sharedMemorySize>>>(d_distances, trainSize, d_trainIndexes, k, d_predictions, d_trainLabels, index, alpha, beta, CLASSES);
    hipDeviceSynchronize();        //forcing synchronous behavior
    double knnElaps = cpuSecond() - knnStart;

    hipMemcpy(distances, d_distances, trainSize * testSize * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(trainIndexes, d_trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(predictions, d_predictions, testSize * sizeof(int), hipMemcpyDeviceToHost);

    //printDataSet(trainData, trainLabels, trainSize);

    //printDistances(distances, testSize, trainSize);

    //printTrainIndexes(trainIndexes, testSize, trainSize);


    //check device results
    int errorCount = checkResult(testLabels, predictions, testSize);

    // kernels dimensions
    unsigned int distDim[4] = {grid.x, grid.y, block.x, block.y};
    unsigned int predDim[4] = {gridDim.x, gridDim.y, blockDim.x, blockDim.y};

    // Write results and device info to file
    writeResultsToFile(testLabels, predictions, errorCount, testSize, "par_results_diabetes.txt", "par_results_diabetes/", trainSize, FEATURES, k, metric, exp, distDim, predDim, workers, alpha, beta, knnDistElaps, knnElaps, sharedMemorySize, maxSharedMemory); 
    //writeDeviceInfo("device_info.txt", device);
    writeAllInfoToFile("all_HW_info.txt", device);

    // Free device memory
    hipFree(d_trainData);
    hipFree(d_testData);
    hipFree(d_distances);
    hipFree(d_trainIndexes);
    hipFree(d_predictions);
    hipFree(d_trainLabels);


    // Free host memory
    free(dataset);
    free(trainData);
    free(trainLabels);
    free(testData);
    free(testLabels);
    free(distances);
    free(trainIndexes);
    free(predictions);


    //reset device
    hipDeviceReset();


    return 0;
}
