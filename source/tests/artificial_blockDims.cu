#include "hip/hip_runtime.h"
#include "../../include/cuda_functions.h"
#include <math.h>

int main(int argc, char** argv) {

    // Selection of best device
    int device = setBestDevice();
    if (device == -1){
        printf("Kernel launch abort\n");
        return -1;
    }

    printf("Executing file: %s\n\n", __FILE__);

    int k = 10; 
    int metric = 1;                                                                             // Euclidean distance
    int exp = 4;                                                                                // Power for Minkowski distance (not used in this case)
    int trainSize = 1000;                                                                       // Size of the training set
    int testSize = 100;                                                                         // Size of the test set
    int mean = 10;                                                                              // Mean value for data generation
    int num_features = 10;                                                                      // Number of features (and classes)
    int num_classes = num_features;                                                             // Number of classes
    int alpha = 2;                                                                              // Default alpha value (regulates the number of threads in a block for the knn kernel)
    int dimx = 2;                                                                               // Initial block dimension
    double exeTimes[5];                                                                         // Execution times for each block dimension

    // Loop over different block dimensions (2, 4, 8, 16, 32 if no argv[1] and argv[2] are provided)
    for(dimx = 2; dimx <= 32; dimx *= 2){

        int dimy = dimx;                                                                                        // Square blocks                      

        // Pointer to memory for data and labels
        double *trainData;
        int *trainLabels;
        double *testData;
        int *testLabels;

        generateData(trainSize, num_features, &trainData, &trainLabels, mean);                                  // Generate training set
        generateData(testSize, num_features, &testData, &testLabels, mean);                                     // Generate test set

        // Host memory allocation
        double *distances = (double *)malloc(trainSize * testSize * sizeof(double));
        int *trainIndexes = (int *)malloc(trainSize * testSize * sizeof(int));
        int *predictions = (int *)malloc(testSize * sizeof(int));

        createTrainIndexes(trainIndexes, testSize, trainSize);                                                  // Create training set indexes for each test set element

        // Pointers to device memory for data, distances, indexes, predictions and labels
        double *d_trainData, *d_testData, *d_distances;
        int *d_trainIndexes, *d_predictions, *d_trainLabels;

        // Device memory allocation for the distances kernel (knnDistances) 
        hipMalloc(&d_trainData, trainSize * num_features * sizeof(double));
        hipMalloc(&d_testData, testSize * num_features * sizeof(double));
        hipMalloc(&d_distances, trainSize * testSize * sizeof(double));

        // Copy data from host to device
        hipMemcpy(d_trainData, trainData, trainSize * num_features * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_testData, testData, testSize * num_features * sizeof(double), hipMemcpyHostToDevice);
        hipMemset(d_distances, 0, trainSize * testSize * sizeof(double));                                      // Initialize distances matrix with 0

        // User defined block dimensions (if provided must be within the maximum number of threads per block)
        if(argc > 2){
            if (atoi(argv[1]) * atoi(argv[2]) <= getMaxThreadsPerBlock(device)){
                dimx = atoi(argv[1]);
                dimy = atoi(argv[2]);
            } else {
                printf("Invalid block dimensions for distances computation. Maximum number of threads per block is %d\n", getMaxThreadsPerBlock(device));
                printf("Using default dimensions: %d x %d\n\n", dimx, dimy);
            }
        }
        dim3 block(dimx, dimy);
        dim3 grid((trainSize + block.x-1)/block.x, (testSize + block.y-1)/block.y);
        
        // Set cache configuration for the kernel -> prefer 48KB L1 cache and 16KB shared memory
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(knnDistances), hipFuncCachePreferL1);

        // Distances computation
        double avgKnnDistElaps = 0.0;                                                                           // Average elapsed time for distances computation
        for(int i = 1; i <= 5; i++){                                                                            // 5 iterations for average time
            double knnDistStart = cpuSecond();
            knnDistances<<< grid, block >>>(d_trainData, d_testData, d_distances, trainSize, testSize, metric, exp, num_features);
            hipDeviceSynchronize();                                                                            //forcing synchronous behavior
            double knnDistElaps = cpuSecond() - knnDistStart;
            avgKnnDistElaps += knnDistElaps;
        }
        avgKnnDistElaps /= 5;

        // Free device memory (to keep available global memory as high as possible for the subsequent memory allocation)
        hipFree(d_trainData);
        hipFree(d_testData);

        // Device memory allocation for the knn kernel
        hipMalloc(&d_trainIndexes, trainSize * testSize * sizeof(int));
        hipMalloc(&d_predictions, testSize * sizeof(int));
        hipMalloc(&d_trainLabels, trainSize * sizeof(int));

        // Copy data from host to device
        hipMemcpy(d_trainIndexes, trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_trainLabels, trainLabels, trainSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_predictions, 0, testSize * sizeof(int));                                                   // Initialize predictions array with 0

        // User defined alpha value (if provided must be within the range [2, 32])
        if(argc > 3){
            if (atoi(argv[3]) >= alpha && atoi(argv[3]) <= 32){         
                alpha = atoi(argv[3]);
            } else {
                printf("Invalid alpha value, alpha must be in range [2, 32]. Using default value: %d\n\n", alpha);
            }
        }

        int beta = 4;                                                                                           // Default beta value
        // User defined beta value (if provided must be within the range [4, 32])
        if(argc > 4){
            if (atoi(argv[4]) >= beta && atoi(argv[4]) <= 32){        
                beta = atoi(argv[4]);
            } else {
                printf("Invalid beta value, beta must be in range [4, 32]. Using default value: %d\n\n", beta);
            }
        }

        int maxSharedMemory = getSharedMemoryPerBlock(device);                                                  // Maximum shared memory per block
        int itemSize = sizeof(double) + sizeof(int);                                                            // Size of each item in shared memory (distance + index)
        int workers = maxSharedMemory/(k * itemSize * (1.5 + 1/alpha));                                         // Default number of threads in a block (maxization of shared memory usage)
        workers = nearestPowerOfTwo(workers);                                                                   // Round the workers number to the nearest power of two
        if (workers > (int)trainSize/(alpha*k)){                                                                // In case of too many workers (small dataset)
            workers = nearestPowerOfTwo((int)trainSize/(alpha*k));                                              // Set new default value  
        }

        // User defined workers value (if provided must be within the range [1, workers])
        if(argc > 5){
                if(atoi(argv[5]) >= 1 && atoi(argv[5]) <= workers){
                workers = atoi(argv[5]);
            } else {
                printf("Invalid workers value. Using default value: %d\n\n", workers);
            }
        }

        dim3 gridDim(testSize, 1, 1);                                                                           // Each block in the grid is responsible for a row of the distances matrix
        dim3 blockDim(workers, 1, 1);                                                                           // Each thread is responsible for a subset of the corresponding row               

        int sharedWorkers = (int)(blockDim.x / alpha);                                                          // Number of workers in shared memory
        int additionalMemory = k * sharedWorkers * (sizeof(double) + sizeof(int));                              // Additional memory needed from shared workers
        int sharedMemorySize = (k * blockDim.x) * (sizeof(double) + sizeof(int)) + additionalMemory;            // Shared memory size
        int index = k * (blockDim.x + sharedWorkers);                                                           // Starting index for trainIndexes in shared memory 

        // KNN computation
        double avgKnnElaps = 0.0;                                                                               // Average elapsed time for knn computation
        for(int i = 1; i <= 5; i++){                                                                            // 5 iterations for average time
            double knnStart = cpuSecond();
            knn<<< gridDim, blockDim, sharedMemorySize>>>(d_distances, trainSize, d_trainIndexes, k, d_predictions, d_trainLabels, index, alpha, beta, num_classes);
            hipDeviceSynchronize();                                                                            // Forcing synchronous behavior
            double knnElaps = cpuSecond() - knnStart;
            avgKnnElaps += knnElaps;
        }
        avgKnnElaps /= 5;
 
        hipMemcpy(predictions, d_predictions, testSize * sizeof(int), hipMemcpyDeviceToHost);                 // Copy predictions from device to host
        int errorCount = checkResult(testLabels, predictions, testSize);                                        // Check the number of errors in the predictions

        // kernels dimensions (needed for the results file)
        unsigned int distDim[4] = {grid.x, grid.y, block.x, block.y};
        unsigned int predDim[4] = {gridDim.x, gridDim.y, blockDim.x, blockDim.y};

        // Print results to file
        appendResultsToFile(errorCount, testSize, "artificial_blockDims_cu.txt", "artificial_blockDims/", trainSize, num_features, k, metric, exp, distDim, predDim, workers, alpha, beta, avgKnnDistElaps, avgKnnElaps, sharedMemorySize, maxSharedMemory, sharedWorkers);

        exeTimes[(int)(log2((double)dimx) - 1)] = avgKnnElaps + avgKnnDistElaps;                                // Store the execution time for the current alpha value

        // Free device memory
        hipFree(d_distances);
        hipFree(d_trainIndexes);
        hipFree(d_predictions);
        hipFree(d_trainLabels);


        // Free host memory
        free(trainData);
        free(trainLabels);
        free(testData);
        free(testLabels);
        free(distances);
        free(trainIndexes);
        free(predictions);

        hipDeviceReset();                                                                                      // Reset the device for the next iteration
    }

    exeTimeToFile("artificial_blockDims_csv.txt", "artificial_blockDims/", exeTimes, 5);

    return 0;
}