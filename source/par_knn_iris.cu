#include "hip/hip_runtime.h"
#include "../include/cuda_functions.h"
#include "../include/iris_functions.h"

int main(int argc, char** argv) {

    // Selection of best device
    int device = setBestDevice();
    if (device == -1){
        printf("Kernel launch abort\n");
        return -1;
    }

    printf("Executing file: %s\n\n", __FILE__);

    int k = 10; 
    int metric = 1;                                                                                             // Euclidean distance
    int exp = 4;                                                                                                // Power for Minkowski distance (not used in this case)
    double exeTimes[5] = {0.0, 0.0, 0.0, 0.0, 0.0};

    IrisData *iris_data;
    int trainSize;

    // Read the Iris dataset
    if (readIrisDataset("../datasets/Iris.csv", &iris_data, &trainSize) != 0) {
        fprintf(stderr, "Error reading Iris dataset\n");
        return 1;
    }

    int testSize = trainSize/3;                                                                                 // Test set size (1/3 of training set)

    // Training set
    double *trainData = (double *)malloc(trainSize * FEATURES * sizeof(double));
    int *trainLabels = (int *)malloc(trainSize * sizeof(int));
    createTrainingSet(iris_data, trainData, trainLabels, trainSize);

    // Test set (1/3 of training set, balanced over classes -> 17,17,16)
    int testDataSize = (trainSize / 3) * FEATURES * sizeof(double);
    int testLabelsSize = (trainSize / 3) * sizeof(int);
    double *testData = (double *)malloc(testDataSize);
    int *testLabels = (int *)malloc(testLabelsSize);
    createTestSet(trainData, testData, trainLabels, testLabels, trainSize);

    // Host memory allocation
    double *distances = (double *)malloc(trainSize * testSize * sizeof(double));
    int *trainIndexes = (int *)malloc(trainSize * testSize * sizeof(int));
    int *predictions = (int *)malloc(testSize * sizeof(int));

    createTrainIndexes(trainIndexes, testSize, trainSize);                                                      // Create training set indexes for each test set element

    // Pointers to device memory for data, distances, indexes, predictions and labels
    double *d_trainData, *d_testData, *d_distances;
    int *d_trainIndexes, *d_predictions, *d_trainLabels;

    // Device memory allocation 
    hipMalloc(&d_trainData, trainSize * FEATURES * sizeof(double));
    hipMalloc(&d_testData, testSize * FEATURES * sizeof(double));
    hipMalloc(&d_distances, trainSize * testSize * sizeof(double));
    hipMalloc(&d_trainIndexes, trainSize * testSize * sizeof(int));
    hipMalloc(&d_predictions, testSize * sizeof(int));
    hipMalloc(&d_trainLabels, trainSize * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_trainData, trainData, trainSize * FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_testData, testData, testSize * FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_trainIndexes, trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_trainLabels, trainLabels, trainSize * sizeof(int), hipMemcpyHostToDevice);
    
    hipMemset(d_distances, 0, trainSize * testSize * sizeof(double));                                              // Initialize distances matrix to 0    
    hipMemset(d_predictions, 0, testSize * sizeof(int));                                                           // Initialize predictions array to 0

    // Set squared maximum dimensions as default for the blocks
    int dimx = (int)sqrt(getMaxThreadsPerBlock(device)); 
    int dimy = dimx;

    // User defined block dimensions (if provided must be whitin the maximum number of threads per block)
    if(argc > 2){
        if (atoi(argv[1]) * atoi(argv[2]) <= getMaxThreadsPerBlock(device)){
            dimx = atoi(argv[1]);
            dimy = atoi(argv[2]);
        } else {
            printf("Invalid block dimensions for distances computation. Maximum number of threads per block is %d\n", getMaxThreadsPerBlock(device));
            printf("Using default dimensions: %d x %d\n\n", dimx, dimy);
        }
    }
    dim3 block(dimx, dimy);
    dim3 grid((trainSize + block.x-1)/block.x, (testSize + block.y-1)/block.y);
    
    // Set cache configuration for the kernel -> prefer 48KB L1 cache and 16KB shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(knnDistances), hipFuncCachePreferL1);

    // Distances computation
    double avgKnnDistElaps = 0.0;                                                                               // Average elapsed time for distances computation                                          
    for(int i = 1; i <= 5; i++){                                                                                // 5 iterations for time average                              
        double knnDistStart = cpuSecond();
        knnDistances<<< grid, block >>>(d_trainData, d_testData, d_distances, trainSize, testSize, metric, exp, FEATURES);
        hipDeviceSynchronize();                                                                                // Forcing synchronous behavior
        exeTimes[i-1] = (cpuSecond() - knnDistStart);
        avgKnnDistElaps += exeTimes[i-1];
    }
    avgKnnDistElaps /= 5;

    
    int alpha = 2;                                                                                              // Default alpha value 
    // User defined alpha value (if provided must be within the range [2, 32])
    if(argc > 3){
        if (atoi(argv[3]) >= alpha && atoi(argv[3]) <= 32){         
            alpha = atoi(argv[3]);
        } else {
            printf("Invalid alpha value, alpha must be in range [2, 32]. Using default value: %d\n\n", alpha);
        }
    }

    int beta = 4;                                                                                           // default beta value
    // User defined beta value (if provided must be within the range [4, 32])
    if(argc > 4){
        if (atoi(argv[4]) >= beta && atoi(argv[4]) <= 32){         
            beta = atoi(argv[4]);
        } else {
            printf("Invalid beta value, beta must be in range [4, 32]. Using default value: %d\n\n", beta);
        }
    }

    int maxSharedMemory = getSharedMemoryPerBlock(device);                                                  // Maximum shared memory per block
    int itemSize = sizeof(double) + sizeof(int);                                                            // Size of each item in shared memory (distance + index)
    int workers = maxSharedMemory/(k * itemSize * (1.5 + 1/alpha));                                         // Default number of threads in a block (maxization of shared memory usage)
    workers = nearestPowerOfTwo(workers);                                                                   // Round the workers number to the nearest power of two
    if (workers > (int)trainSize/(alpha*k)){                                                                // In case of too many workers (small dataset)
        workers = nearestPowerOfTwo((int)trainSize/(alpha*k));                                              // Set new default value  
    }

    // User defined workers number (if provided must be within the range [1, workers])
    if(argc > 5){
        if (atoi(argv[5]) < workers && atoi(argv[5]) >= 1){
            workers = atoi(argv[5]);
        } else {
            printf("Invalid workers value. Using default value: %d\n\n", workers);
        }
    }

    dim3 gridDim(testSize, 1, 1);                                                                           // Each block in the grid is responsible for a row of the distances matrix
    dim3 blockDim(workers, 1, 1);                                                                           // Each thread is responsible for a subset of the corresponding row       

    int sharedWorkers = (int)(blockDim.x / alpha);                                                          // Number of workers in shared memory
    int additionalMemory = k * sharedWorkers * (sizeof(double) + sizeof(int));                              // Additional memory needed from shared workers
    int sharedMemorySize = (k * blockDim.x) * (sizeof(double) + sizeof(int)) + additionalMemory;            // Shared memory size
    int index = k * (blockDim.x + sharedWorkers);                                                           // Starting index for trainIndexes in shared memory 

    // KNN computation
    double avgKnnElaps = 0.0;                                                                               // Average elapsed time for KNN computation        
    for(int i = 1; i <= 5; i++){                                                                            // 5 iterations for time average
        double knnStart = cpuSecond();
        knn<<< gridDim, blockDim, sharedMemorySize>>>(d_distances, trainSize, d_trainIndexes, k, d_predictions, d_trainLabels, index, alpha, beta, CLASSES);
        hipDeviceSynchronize();                                                                            // Forcing synchronous behavior
        double knnElaps = (cpuSecond() - knnStart);
        exeTimes[i-1] += knnElaps;
        avgKnnElaps += knnElaps;
    }
    avgKnnElaps /= 5;

    double exeTime = avgKnnDistElaps + avgKnnElaps;                                                         // Total execution time

    hipMemcpy(predictions, d_predictions, testSize * sizeof(int), hipMemcpyDeviceToHost);                 // Copy predictions from device to host
    int errorCount = checkResult(testLabels, predictions, testSize);                                        // Check the number of errors in the predictions

    // kernels dimensions 
    unsigned int distDim[4] = {grid.x, grid.y, block.x, block.y};
    unsigned int predDim[4] = {gridDim.x, gridDim.y, blockDim.x, blockDim.y};

    // Print results to file
    appendRunStatsToFile("iris_cu.txt", "iris/", exeTimes, 5);
    appendResultsToFile(errorCount, testSize, "iris_cu.txt", "iris/", trainSize, FEATURES, k, metric, exp, distDim, predDim, workers, alpha, beta, avgKnnDistElaps, avgKnnElaps, sharedMemorySize, maxSharedMemory, sharedWorkers);

    // Free device memory
    hipFree(d_trainData);
    hipFree(d_testData);
    hipFree(d_distances);
    hipFree(d_trainIndexes);
    hipFree(d_predictions);
    hipFree(d_trainLabels);


    // Free host memory
    free(iris_data);
    free(trainData);
    free(trainLabels);
    free(testData);
    free(testLabels);
    free(distances);
    free(trainIndexes);
    free(predictions);

    hipDeviceReset();                                                                                       // Reset the device for the next iteration

    exeTimeToFile("iris_csv.txt", "iris/", &exeTime, 1);                                                     // Write execution time to csv file                                   

    return 0;
}
