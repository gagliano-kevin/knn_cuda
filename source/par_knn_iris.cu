#include "hip/hip_runtime.h"
#include "../include/cuda_functions.h"
#include "../include/iris_functions.h"

int main(int argc, char** argv) {

    // Selection of best device
    int device = setBestDevice();
    if (device == -1){
        printf("Kernel launch abort\n");
        return -1;
    }

    printf("Executing file: %s\n\n", __FILE__);

    int k = 5; 
    int metric = 1; // Metric distance
    int exp = 4; // Power for Minkowski distance

    IrisData *iris_data;
    int trainSize;

    // Read the Iris dataset
    if (readIrisDataset("../datasets/Iris.csv", &iris_data, &trainSize) != 0) {
        fprintf(stderr, "Error reading Iris dataset\n");
        return 1;
    }

    int testSize = trainSize/3;

    double *trainData = (double *)malloc(trainSize * FEATURES * sizeof(double));
    int *trainLabels = (int *)malloc(trainSize * sizeof(int));

    createTrainingSet(iris_data, trainData, trainLabels, trainSize);

    // Test set (1/3 of training set, balanced over classes -> 17,17,16)
    size_t testDataSize = (trainSize / 3) * FEATURES * sizeof(double);
    size_t testLabelsSize = (trainSize / 3) * sizeof(int);
    
    double *testData = (double *)malloc(testDataSize);
    int *testLabels = (int *)malloc(testLabelsSize);
    
    createTestSet(trainData, testData, trainLabels, testLabels, trainSize);

    double *distances = (double *)malloc(trainSize * testSize * sizeof(double));

    int *trainIndexes = (int *)malloc(trainSize * testSize * sizeof(int));
    int *predictions = (int *)malloc(testSize * sizeof(int));


    createTrainIndexes(trainIndexes, testSize, trainSize);

    // device
    double *d_trainData, *d_testData, *d_distances;
    int *d_trainIndexes, *d_predictions, *d_trainLabels;

    hipMalloc(&d_trainData, trainSize * FEATURES * sizeof(double));
    hipMalloc(&d_testData, testSize * FEATURES * sizeof(double));
    hipMalloc(&d_distances, trainSize * testSize * sizeof(double));
    hipMalloc(&d_trainIndexes, trainSize * testSize * sizeof(int));
    hipMalloc(&d_predictions, testSize * sizeof(int));
    hipMalloc(&d_trainLabels, trainSize * sizeof(int));

    hipMemcpy(d_trainData, trainData, trainSize * FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_testData, testData, testSize * FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_trainIndexes, trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_trainLabels, trainLabels, trainSize * sizeof(int), hipMemcpyHostToDevice);
    

    // Set squared maximum dimensions as default
    int dimx = (int)sqrt(getMaxThreadsPerBlock(device)); 
    int dimy = dimx;

    if(argc > 2){
        if (atoi(argv[1]) * atoi(argv[2]) <= getMaxThreadsPerBlock(device)){
            dimx = atoi(argv[1]);
            dimy = atoi(argv[2]);
        } else {
            printf("Invalid block dimensions for distances computation. Maximum number of threads per block is %d\n", getMaxThreadsPerBlock(device));
            printf("Using default dimensions: %d x %d\n\n", dimx, dimy);
        }
    }
    dim3 block(dimx, dimy);
    dim3 grid((trainSize + block.x-1)/block.x, (testSize + block.y-1)/block.y);

    hipMemset(d_distances, 0, trainSize * testSize * sizeof(double)); // initialize distances matrix with 0
    
    // Set cache configuration for the kernel -> prefer 48KB L1 cache and 16KB shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(knnDistances), hipFuncCachePreferL1);

    double avgKnnDistElaps = 0.0;
    for(int i = 1; i <= 5; i++){
        double knnDistStart = cpuSecond();
        knnDistances<<< grid, block >>>(d_trainData, d_testData, d_distances, trainSize, testSize, metric, exp, FEATURES);
        hipDeviceSynchronize();        //forcing synchronous behavior
        avgKnnDistElaps += (cpuSecond() - knnDistStart);
    }
    avgKnnDistElaps /= 5;

    
    int alpha = 2;  // default
    if(argc > 3 && atoi(argv[3]) <= 32){        // alpha limited up to 32
        if (atoi(argv[3]) >= alpha){
            alpha = atoi(argv[3]);
        } else {
            printf("Invalid alpha value, alpha must be in range [2, 32]. Using default value: %d\n\n", alpha);
        }
    }

    int beta = 4;   // default
    if(argc > 4){
        if (atoi(argv[4]) >= beta && atoi(argv[4]) <= 32){         // beta limited up to 32
            beta = atoi(argv[4]);
        } else {
            printf("Invalid beta value, beta must be in range [4, 32]. Using default value: %d\n\n", beta);
        }
    }

    int maxSharedMemory = getSharedMemoryPerBlock(device);
    int itemSize = sizeof(double) + sizeof(int);
    int workers = maxSharedMemory/(k * itemSize * (1.5 + 1/alpha));       // default (maxization of shared memory usage)
    workers = nearestPowerOfTwo(workers);
    if (workers > (int)trainSize/(alpha*k)){
        workers = nearestPowerOfTwo((int)trainSize/(alpha*k));           // new default in case of too many workers (small dataset)
    }

    if(argc > 5){
        if (atoi(argv[5]) < workers && atoi(argv[5]) >= 1){
            workers = atoi(argv[5]);
        } else {
            printf("Invalid workers value. Using default value: %d\n\n", workers);
        }
    }

    dim3 gridDim(testSize, 1, 1);   // each thread block is responsible for a row of the distances matrix
    dim3 blockDim(workers, 1, 1);

    int sharedWorkers = (int)(blockDim.x / alpha);
    int additionalMemory = k * sharedWorkers * (sizeof(double) + sizeof(int));  // blockDim.x/alpha is the number of workers in 2^ iteration (first in shared memory)

    int sharedMemorySize = (k * blockDim.x) * (sizeof(double) + sizeof(int)) + additionalMemory; 

    int index = k * (blockDim.x + sharedWorkers); // starting index for trainIndexes in shared memory 

    double avgKnnElaps = 0.0;
    for(int i = 1; i <= 5; i++){
        double knnStart = cpuSecond();
        knn<<< gridDim, blockDim, sharedMemorySize>>>(d_distances, trainSize, d_trainIndexes, k, d_predictions, d_trainLabels, index, alpha, beta, CLASSES);
        hipDeviceSynchronize();        //forcing synchronous behavior
        avgKnnElaps += (cpuSecond() - knnStart);
    }
    avgKnnElaps /= 5;

    double exeTime = avgKnnDistElaps + avgKnnElaps;

    hipMemcpy(distances, d_distances, trainSize * testSize * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(trainIndexes, d_trainIndexes, trainSize * testSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(predictions, d_predictions, testSize * sizeof(int), hipMemcpyDeviceToHost);

    //printDataSet(trainData, trainLabels, trainSize);

    //printDistances(distances, testSize, trainSize);

    //printTrainIndexes(trainIndexes, testSize, trainSize);


    //check device results
    int errorCount = checkResult(testLabels, predictions, testSize);

    // kernels dimensions
    unsigned int distDim[4] = {grid.x, grid.y, block.x, block.y};
    unsigned int predDim[4] = {gridDim.x, gridDim.y, blockDim.x, blockDim.y};

    appendResultsToFile(errorCount, testSize, "iris_cu.txt", "iris/", trainSize, FEATURES, k, metric, exp, distDim, predDim, workers, alpha, beta, avgKnnDistElaps, avgKnnElaps, sharedMemorySize, maxSharedMemory, sharedWorkers);

    // Write results and device info to file
    // writeResultsToFile(testLabels, predictions, errorCount, testSize, "par_results_iris.txt", "par_results_iris/", trainSize, FEATURES, k, metric, exp, distDim, predDim, workers, alpha, beta, avgKnnDistElaps, avgKnnElaps, sharedMemorySize, maxSharedMemory, sharedWorkers); 
    //writeDeviceInfo("device_info.txt", device);
    //writeAllInfoToFile("all_HW_info.txt", device);

    // Free device memory
    hipFree(d_trainData);
    hipFree(d_testData);
    hipFree(d_distances);
    hipFree(d_trainIndexes);
    hipFree(d_predictions);
    hipFree(d_trainLabels);


    // Free host memory
    free(iris_data);
    free(trainData);
    free(trainLabels);
    free(testData);
    free(testLabels);
    free(distances);
    free(trainIndexes);
    free(predictions);


    //reset device
    hipDeviceReset();

    exeTimeToFile("iris_csv.txt", "iris/", &exeTime, 1);


    return 0;
}
